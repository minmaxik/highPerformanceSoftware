
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello() {
  printf("Hello from thread # %d (block %d)\n", threadIdx.x, blockIdx.x);
}

int main() {
  hello<<<2,4>>>();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel failed: %s", hipGetErrorString(err));
  }
  
  hipDeviceSynchronize();
  
  return 0;
}
